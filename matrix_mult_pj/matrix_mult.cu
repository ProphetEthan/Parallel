#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <chrono>
#include <string>
#include <iomanip>
#include <sstream>

#define BLOCKSIZE 16

#define SAFE(call)                                                         \
            do{                                                                      \
                 hipError_t err = call;                                             \
                 if(err != hipSuccess)                                              \
                 {                                                                   \
                        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                         __FILE__, __LINE__, hipGetErrorString( err) );             \
                         exit(1);                                                    \
                 }                                                                   \
               } while (0)                                                           \

__global__ void MatrixMultiplication(float *MA, float *MB, float *Res, int r1, int c1, int c2, int cut_size, int id) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < cut_size && col < c2) {
        float sum = 0.0;
        for (int k = 0; k < c1; ++k) {
            sum += MA[row * c1 + k] * MB[k * c2 + col];
        }
        Res[row * c2 + col] = sum;
    }
}

int IntializingMatrixVectors(float **MA, float **MB, float **ResultM, int R1, int C1, int R2, int C2){
    float *TempMA, *TempResultM, *TempMB;
    int Status = 1;

    TempMA = (float *)malloc(R1 * C1 * sizeof(float));
    if(TempMA == NULL) Status = 0;

    TempMB = (float *)malloc(R2 * C2 * sizeof(float));
    if(TempMB == NULL) Status = 0;

    TempResultM = (float *)malloc(R1 * C2 * sizeof(float));
    if(TempResultM == NULL) Status = 0;

    int limit = 10;

    for(int i = 0; i < R1 * C1; i++)
        TempMA[i] = (float)rand() / (float)(RAND_MAX / limit);

    for(int i = 0; i < R2 * C2; i++)
        TempMB[i] = (float)rand() / (float)(RAND_MAX / limit);

    for(int i = 0; i < R1 * C2; i++)
        TempResultM[i] = 0.0f;

    *MA = TempMA;
    *MB = TempMB;
    *ResultM = TempResultM;

    return Status;
}

int CheckCudaDevice(int id) {
    int DeviceCount, Device;
    struct hipDeviceProp_t Properties;

    hipGetDeviceCount(&DeviceCount);
    if(DeviceCount >= 1) {
        hipGetDevice(&Device);
        hipGetDeviceProperties(&Properties, Device);
        printf("Processor with rank %d has the Device by name %s and computation is done on this device \n", id, Properties.name);
    } else {
        printf("Processor with rank %d found no CUDA device.\n", id);
        return 0;
    }
    return DeviceCount;
}

void printMatrix(float *M, int R, int C, const char *filename) {
    FILE *file = fopen(filename, "a");
    if (file == NULL) {
        perror("Error opening file");
        return;
    }

    for (int i = 0; i < R; i++) {
        for (int j = 0; j < C; j++) {
            fprintf(file, "%f", M[i * C + j]);
            if (j < C - 1) {
                fprintf(file, ",");
            }
        }
        fprintf(file, "\n");
    }

    fclose(file);
}

float abs(float a, float b) {
    return (a >= b) ? a - b : b - a;
}

std::string getCurrentTime() {
    auto now = std::chrono::system_clock::now();
    std::time_t now_time_t = std::chrono::system_clock::to_time_t(now);
    std::tm now_tm;
    localtime_r(&now_time_t, &now_tm);
    std::ostringstream oss;
    oss << std::put_time(&now_tm, "%Y-%m-%d %H:%M:%S");
    return oss.str();
}

double calculate_mult_time(float *MA, float *MB, float *RM, int R1, int C1, int R2, int C2) {
    double s_time = MPI_Wtime();
    for (int i = 0; i < R1; i++) {
        for (int j = 0; j < C2; j++) {
            float sum = 0.0;
            for (int k = 0; k < R2; k++) sum += MA[i * C1 + k] * MB[k * C2 + j];
            RM[i * C2 + j] = sum;
        }
    }
    double e_time = MPI_Wtime();
    return e_time - s_time;
}

int main(int argc, char **argv) {
    int id, num_proc;
    int Root = 0, Status = 1;
    float *MatrixA, *MatrixB, *ResultM, *RM;
    float *Cut_MatrixA, *Cut_ResultM;
    float *CudaMA, *CudaRM, *CudaMB;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &id);
    MPI_Comm_size(MPI_COMM_WORLD, &num_proc);

    if (argc < 4) {
        if (id == Root) printf("Invalid input args\n");
        MPI_Finalize();
        exit(-1);
    }

    int R1 = atoi(argv[1]), C1 = atoi(argv[2]), R2 = atoi(argv[3]), C2 = atoi(argv[4]);

    if (C1 != R2) {
        if (id == Root) std::cout << "Entered wrong input, Number of columns of matrix 1 should be equal to number of rows of matrix 2 " << std::endl;
        MPI_Finalize();
        exit(-1);
    }

    if (R1 < num_proc) {
        if (id == Root) std::cout << "Given number of Rows of the matrix should be more than number of processors" << std::endl;
        MPI_Finalize();
        exit(-1);
    }

    if (R1 % num_proc != 0) {
        if (id == Root) std::cout << "The Rows of the matrix cannot be distributed evenly among processors " << std::endl;
        MPI_Finalize();
        exit(-1);
    }

    if (id == Root) Status = IntializingMatrixVectors(&MatrixA, &MatrixB, &ResultM, R1, C1, R2, C2);

    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Bcast(&Status, 1, MPI_INT, Root, MPI_COMM_WORLD);

    if (id != Root) MatrixB = (float *)malloc(R2 * C2 * sizeof(float));

    MPI_Bcast(MatrixB, R2 * C2, MPI_FLOAT, Root, MPI_COMM_WORLD);

    int cut_size = R1 / num_proc;
    Cut_MatrixA = (float *)malloc(cut_size * C1 * sizeof(float));
    if (Cut_MatrixA == NULL) Status = 0;

    Cut_ResultM = (float *)malloc(R1 * C2 * sizeof(float) / num_proc);
    if (Cut_ResultM == NULL) Status = 0;

    ResultM = (float *)malloc(R1 * C2 * sizeof(float));
    if (ResultM == NULL) Status = 0;

    MPI_Scatter(MatrixA, cut_size * C1, MPI_FLOAT, Cut_MatrixA, cut_size * C1, MPI_FLOAT, Root, MPI_COMM_WORLD);

    if (CheckCudaDevice(id) == 0) {
        std::cout << "No CUDA device is found! Using CPU only" << std::endl;

        for (int i = 0; i < cut_size; i++) {
            Cut_ResultM[i] = 0;
            int indexs = i * C1;
            for (int col = 0; col < C1; col++) {
                Cut_ResultM[i] += (Cut_MatrixA[indexs] * MatrixB[col]);
                indexs++;
            }
        }
    } else {
        hipSetDevice(id);
        SAFE(hipMalloc((void **)&CudaMA, cut_size * C1 * sizeof(float)));
        SAFE(hipMalloc((void **)&CudaMB, R2 * C2 * sizeof(float)));
        SAFE(hipMalloc((void **)&CudaRM, R1 * C2 * sizeof(float) / num_proc));

        SAFE(hipMemcpy((void *)CudaMA, (void *)Cut_MatrixA, cut_size * C1 * sizeof(float), hipMemcpyHostToDevice));
        SAFE(hipMemcpy((void *)CudaMB, (void *)MatrixB, R2 * C2 * sizeof(float), hipMemcpyHostToDevice));

        double s_time = MPI_Wtime();

        dim3 blockSize(BLOCKSIZE, BLOCKSIZE);
        dim3 gridSize((C2 + BLOCKSIZE - 1) / BLOCKSIZE, (cut_size + BLOCKSIZE - 1) / BLOCKSIZE);
        MatrixMultiplication<<<gridSize, blockSize>>>(CudaMA, CudaMB, CudaRM, R1, C1, C2, cut_size, id);
        SAFE(hipDeviceSynchronize());

        SAFE(hipMemcpy(Cut_ResultM, CudaRM, cut_size * C2 * sizeof(float), hipMemcpyDeviceToHost));

        double e_time = MPI_Wtime();

        MPI_Gather(Cut_ResultM, cut_size * C2, MPI_FLOAT, ResultM, cut_size * C2, MPI_FLOAT, Root, MPI_COMM_WORLD);

        if (id == Root) {
            RM = (float *)malloc(R1 * C2 * sizeof(float));
            double cpu_time = calculate_mult_time(MatrixA, MatrixB, RM, R1, C1, R2, C2);

            bool IfTrue = true;
            for (int i = 0; i < R1 * C2; i++) {
                if (abs(RM[i], ResultM[i]) > 0.01) {
                    IfTrue = false;
                    printf("%f, %f", RM[i], ResultM[i]);
                    break;
                }
            }
            printf("Result are %s\n", IfTrue ? "true" : "false");

            // FILE *file = fopen("./matrix_record.txt", "a");
            // std::string currentTime = getCurrentTime();
            // fprintf(file, "Current Time: %s\n", currentTime.c_str());
            // fprintf(file, "Matrix A:\n");
            // printMatrix(MatrixA, R1, C1, "./matrix_record.txt");
            // fflush(stdout); 
            // fprintf(file, "Matrix B:\n");
            // printMatrix(MatrixB, R2, C2, "./matrix_record.txt");
            // fflush(stdout); 
            // fprintf(file, "CPU Matrix R:\n");
            // printMatrix(RM, R1, C2, "./matrix_record.txt");
            // fflush(stdout); 
            // fprintf(file, "CUDA Matrix R:\n");
            // printMatrix(ResultM, R1, C2, "./matrix_record.txt");
            // fflush(stdout); 
            std::cout << "Time for CPU:" << cpu_time << std::endl;
            std::cout << "Time for GPU:" << e_time - s_time << std::endl;
            std::cout << "Speed up rate:" << cpu_time / (e_time - s_time) << std::endl;
            // fclose(file);
        }
    }

    free(Cut_MatrixA);
    free(MatrixB);
    free(Cut_ResultM);

    SAFE(hipFree(CudaMA));
    SAFE(hipFree(CudaMB));
    SAFE(hipFree(CudaRM));

    MPI_Finalize();
    return 0;
}
